#include "hip/hip_runtime.h"
#include <windows.h>
#include <thread>
#include <iostream>
#include <unordered_map>
#include <tbb/tbb.h>
#include <stdio.h>
#include <math.h>
#include "../CellGrowth_CUDA/Dependencies/glew/glew.h"
#include "../CellGrowth_CUDA/Dependencies/freeglut/freeglut.h"
#include "hip/hip_runtime.h"
#include ""
#include "angle_bracket_replace_macro.h"

using namespace std;
using namespace tbb;

//global variables
const int WIDTH = 1024, HEIGHT = 768;
bool flag = true;

/*
*	data model
*/
int cell[WIDTH][HEIGHT] = { {0} };
int medDir[WIDTH][HEIGHT] = { {0} };

int host_cell[WIDTH*HEIGHT] = { 0 };
int host_medDir[WIDTH*HEIGHT] = { 0 };

/*
*	viewer methods
*/
void init() {
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glLoadIdentity();
	//gluOrtho2D(-0.5f, WIDTH - 0.5f, -0.5f, HEIGHT - 0.5f);
	gluOrtho2D(-0.5f, 400 - 0.5f, -0.5f, 300 - 0.5f);
}

void draw(GLfloat red, GLfloat green, GLfloat blue, int x, int y) {
	glPointSize(3.0f);
	glColor3f(red, green, blue);
	glBegin(GL_POINTS);
	glVertex2i(x, y);
	glEnd();
}

void update(int value) {
	glutPostRedisplay();
	glutTimerFunc(10.00, update, 0);
}


/*
*	controller methods
*/
void setupCUDA(int x, int y, int m) {
	srand(time(NULL));
	rand(); rand(); rand();
	int h = (m * x) + 2;
	int w = (m * y) + 2;

	for (int i = 0; i < 50; i++) {
		int center_x;
		int center_y;
		do {
			center_x = ((h - x) + 1) + rand() / (RAND_MAX / ((h - 2) - ((h - x) + 1)));
			center_y = ((w - y) + 1) + rand() / (RAND_MAX / ((w - 2) - ((w - y) + 1)));
		} while (host_cell[center_x * HEIGHT + center_y] != 0);
		host_cell[center_x * HEIGHT + center_y] = 9;

		host_cell[center_x * HEIGHT + center_y + 1] = 3;
		host_cell[(center_x - 1) * HEIGHT + center_y + 1] = 3;
		host_cell[(center_x - 1) * HEIGHT + center_y] = 3;
		host_cell[(center_x - 1) * HEIGHT + center_y - 1] = 3;
		host_cell[center_x * HEIGHT + center_y - 1] = 3;
		host_cell[(center_x + 1) * HEIGHT + center_y - 1] = 3;
		host_cell[(center_x + 1) * HEIGHT + center_y] = 3;
		host_cell[(center_x + 1) * HEIGHT + center_y + 1] = 3;

		host_medDir[center_x * HEIGHT + center_y + 1] = 1;	//up
		host_medDir[(center_x - 1) * HEIGHT + center_y + 1] = 2; //left_up
		host_medDir[(center_x - 1) * HEIGHT + center_y] = 3;	//left;
		host_medDir[(center_x - 1) * HEIGHT + center_y - 1] = 4; //left_down;
		host_medDir[center_x * HEIGHT + center_y - 1] = 5; //down;
		host_medDir[(center_x + 1) * HEIGHT + center_y - 1] = 6;  //right_down
		host_medDir[(center_x + 1) * HEIGHT + center_y] = 7; //right
		host_medDir[(center_x + 1) * HEIGHT + center_y + 1] = 8;  //right_up;
	}
	for (int i = (h - x); i < h; i++) {
		for (int j = (w - y); j < w; j++) {
			//Initialize each pixel with an arbitry alive/dead value.
			if (host_cell[i * HEIGHT + j] != 3) {
				host_cell[i * HEIGHT + j] = ((rand() % 4) == 0) ? 1 : 2;
			}
		}
	}
}

__device__ void medCellMoveCUDA(int o_x, int o_y, int t_x, int t_y, int direction, int* dev_cell, int* dev_medDir, int num_per_row) {
	int typeTemp = typeTemp = dev_cell[t_x * num_per_row + t_y];
	dev_cell[o_x * num_per_row + o_y] = typeTemp;
	dev_cell[t_x * num_per_row + t_y] = 3;
	dev_medDir[o_x * num_per_row + o_y] = 0;
	dev_medDir[t_x * num_per_row + t_y] = direction;
}

//Check status of individual cell and apply the game rules.
__device__ void checkStatusCUDA(int* dev_cell, int cur_row, int cur_column, int num_per_row, int* state) {
	int cancerNeighbours = 0;
	int liveNeighbours = 0;
	int medNeighbours = 0;

	int upper_row = cur_row - 1 < 0 ? 0 : cur_row - 1;
	int lower_row = cur_row + 1 > 1023 ? 1023 : cur_row + 1;
	int left_column = cur_column - 1 < 0 ? 0 : cur_column - 1;
	int right_column = cur_column + 1 > 767 ? 767 : cur_column + 1;
	for (int i = upper_row; i <= lower_row; i++) {
		
		if (dev_cell[i*num_per_row + left_column] == 1 && left_column!=cur_column) {
			cancerNeighbours++;
		}
		else if (dev_cell[i*num_per_row + left_column] == 2 && left_column != cur_column) {
			liveNeighbours++;
		}
		else if (dev_cell[i*num_per_row + left_column] == 3 && left_column != cur_column) {
			medNeighbours++;
		}
		if (dev_cell[i*num_per_row + right_column] == 1 && right_column!= cur_column) {
			cancerNeighbours++;
		}
		else if (dev_cell[i*num_per_row + right_column] == 2 && right_column != cur_column) {
			liveNeighbours++;
		}
		else if (dev_cell[i*num_per_row + right_column] == 3 && right_column != cur_column) {
			medNeighbours++;
		}
	}
	if (dev_cell[cur_row*num_per_row + cur_column] != 0) {
		if (dev_cell[upper_row*num_per_row + cur_column] == 1 && upper_row!=cur_row) {
			cancerNeighbours++;
		}
		else if (dev_cell[upper_row*num_per_row + cur_column] == 2 && upper_row != cur_row) {
			liveNeighbours++;
		}
		else if (dev_cell[upper_row*num_per_row + cur_column] == 3 && upper_row != cur_row) {
			medNeighbours++;
		}

		if (dev_cell[lower_row*num_per_row + cur_column] == 1 && lower_row!=cur_row) {
			cancerNeighbours++;
		}
		else if (dev_cell[lower_row*num_per_row + cur_column] == 2 && lower_row != cur_row) {
			liveNeighbours++;
		}
		else if (dev_cell[lower_row*num_per_row + cur_column] == 3 && lower_row != cur_row) {
			medNeighbours++;
		}

		if (dev_cell[cur_row*num_per_row + cur_column] == 1 && medNeighbours >= 3) {
			*state = 2;
		}
		else if (dev_cell[cur_row*num_per_row + cur_column] == 2 && cancerNeighbours >= 5) {
			*state = 1;
		}
		else
			*state = dev_cell[cur_row*num_per_row + cur_column];
	}
	else {
		if (liveNeighbours == 3) {
			*state = 2;
		}
	}
}

void medMultiInjectionCUDA() {
	//variable indicating how many groups of medicine should be injected at a time
	int num_medicines_group = rand() % (15) + 1;
	int inject_area_width = 400;
	int inject_area_height = 300;
	int h = (1 * inject_area_width) + 2;
	int w = (1 * inject_area_height) + 2;
	bool ifInjected[WIDTH*HEIGHT] = { false };

	parallel_for(blocked_range<size_t>(0, num_medicines_group), [&](blocked_range<size_t> & a) {
		for (size_t i = a.begin(); i != a.end(); ++i) {
			//variable indicating the number of medicine cell which should be line up at the border (not including 4 corner)
			//also variable indicating how many cells should the medicine cell be placed far away from center 
			int num_medicine_factor = rand() % (15) + 1;
			//variable indicating the total number of medicine for a single group of medicine cells in one step of iteration
			int num_medicine = num_medicine_factor * 4 + 4;

			int center_x;
			int center_y;
			do {
				center_x = ((h - inject_area_width) + num_medicine_factor) + rand() / (RAND_MAX / ((h - 2) - ((h - inject_area_width) + num_medicine_factor)));
				center_y = ((w - inject_area_height) + num_medicine_factor) + rand() / (RAND_MAX / ((w - 2) - ((w - inject_area_height) + num_medicine_factor)));
			} while (ifInjected[center_x*HEIGHT + center_y] == true);
			ifInjected[center_x*HEIGHT + center_y] = true;

			parallel_for(blocked_range<size_t>(center_x - (int)num_medicine_factor / 2, center_x + (int)num_medicine_factor / 2 + 1), [&](blocked_range<size_t>& r) {
				for (int j = r.begin(); j != r.end(); ++j) {
					host_cell[j*HEIGHT + center_y + num_medicine_factor] = 3;
					host_cell[j*HEIGHT + center_y - num_medicine_factor] = 3;
					host_medDir[j*HEIGHT + center_y + num_medicine_factor] = 1;	//up
					host_medDir[j*HEIGHT + center_y - num_medicine_factor] = 5; //down;
				}
			});

			parallel_for(blocked_range<size_t>(center_y - (int)num_medicine_factor / 2, center_y + (int)num_medicine_factor / 2 + 1), [&](blocked_range<size_t>& r) {
				for (int j = r.begin(); j != r.end(); ++j) {
					host_cell[(center_x - num_medicine_factor)*HEIGHT + j] = 3;
					host_cell[(center_x + num_medicine_factor)*HEIGHT + j] = 3;
					host_medDir[(center_x - num_medicine_factor)*HEIGHT + j] = 3;	//left
					host_medDir[(center_x + num_medicine_factor)*HEIGHT + j] = 7; //right;
				}
			});

			host_cell[(center_x - num_medicine_factor)*HEIGHT + center_y + num_medicine_factor] = 3;
			host_cell[(center_x - num_medicine_factor)*HEIGHT + center_y - num_medicine_factor] = 3;
			host_cell[(center_x + num_medicine_factor)*HEIGHT + center_y - num_medicine_factor] = 3;
			host_cell[(center_x + num_medicine_factor)*HEIGHT + center_y + num_medicine_factor] = 3;
			host_medDir[(center_x - num_medicine_factor)*HEIGHT + center_y + num_medicine_factor] = 2; //left_up
			host_medDir[(center_x - num_medicine_factor)*HEIGHT + center_y - num_medicine_factor] = 4; //left_down;
			host_medDir[(center_x + num_medicine_factor)*HEIGHT + center_y - num_medicine_factor] = 6;  //right_down
			host_medDir[(center_x + num_medicine_factor)*HEIGHT + center_y + num_medicine_factor] = 8;  //right_up;
		}
	});
}

void medInjectionCUDA(GLdouble worldX, GLdouble worldY) {
	int center_x = (int)worldX;
	int center_y = (int)worldY;

	host_cell[center_x * HEIGHT + center_y + 1] = 3;
	host_cell[(center_x - 1) * HEIGHT + center_y + 1] = 3;
	host_cell[(center_x - 1) * HEIGHT + center_y] = 3;
	host_cell[(center_x - 1) * HEIGHT + center_y - 1] = 3;
	host_cell[center_x *HEIGHT + center_y - 1] = 3;
	host_cell[(center_x + 1) * HEIGHT + center_y - 1] = 3;
	host_cell[(center_x + 1) * HEIGHT + center_y] = 3;
	host_cell[(center_x + 1) * HEIGHT + center_y + 1] = 3;

	host_medDir[center_x * HEIGHT + center_y + 1] = 1;	//up
	host_medDir[(center_x - 1) *HEIGHT + center_y + 1] = 2; //left_up
	host_medDir[(center_x - 1) * HEIGHT + center_y] = 3;	//left;
	host_medDir[(center_x - 1) * HEIGHT + center_y - 1] = 4; //left_down;
	host_medDir[center_x * HEIGHT + center_y - 1] = 5; //down;
	host_medDir[(center_x + 1) * HEIGHT + center_y - 1] = 6;  //right_down
	host_medDir[(center_x + 1) * HEIGHT + center_y] = 7; //right
	host_medDir[(center_x + 1) * HEIGHT + center_y + 1] = 8;  //right_up;
}

void OnMouseClick(int button, int state, int x, int y)
{
	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP)
	{
		GLint viewport[4]; //var to hold the viewport info
		GLdouble modelview[16]; //var to hold the modelview info
		GLdouble projection[16]; //var to hold the projection matrix info
		GLfloat winX, winY, winZ; //variables to hold screen x,y,z coordinates
		GLdouble worldX, worldY, worldZ; //variables to hold world x,y,z coordinates
		glGetDoublev(GL_MODELVIEW_MATRIX, modelview); //get the modelview info
		glGetDoublev(GL_PROJECTION_MATRIX, projection); //get the projection matrix info
		glGetIntegerv(GL_VIEWPORT, viewport); //get the viewport info
		winX = (float)x;
		winY = (float)viewport[3] - (float)y;
		winZ = 0;
		//get the world coordinates from the screen coordinates
		gluUnProject(winX, winY, winZ, modelview, projection, viewport, &worldX, &worldY, &worldZ);
		medInjectionCUDA(worldX, worldY);
		medMultiInjectionCUDA();
		//cout << "worldX: " << worldX << "worldY: " << worldY << endl;
	}
}

//Display individual pixels.
__global__ void cellHandle(int *dev_cell, int* dev_medDir, int num_rows, int num_per_row) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int cur_row = (int) idx / num_per_row;
	int cur_column = idx % num_per_row;

	int state;
	checkStatusCUDA(dev_cell, cur_row, cur_column, num_per_row, &state);

	if (state == 2 && dev_cell[cur_row*num_per_row + cur_column] == 1) {
		if (cur_column + 1 < HEIGHT - 5 && dev_cell[cur_row*num_per_row + cur_column + 1] == 3) {
			dev_cell[cur_row*num_per_row + cur_column + 1] = 2;
			dev_medDir[cur_row*num_per_row + cur_column + 1] = 0;
		}
		if (cur_row - 1 >= 5 && cur_column + 1 < HEIGHT - 5 && dev_cell[(cur_row - 1)*num_per_row + cur_column + 1] == 3) {
			dev_cell[(cur_row - 1)*num_per_row + cur_column + 1] = 2;
			dev_medDir[(cur_row - 1)*num_per_row + cur_column + 1] = 0;
		}
		if (cur_row - 1 >= 5 && dev_cell[(cur_row - 1)*num_per_row + cur_column] == 3) {
			dev_cell[(cur_row - 1)*num_per_row + cur_column] = 2;
			dev_medDir[(cur_row - 1)*num_per_row + cur_column] = 0;
		}
		if (cur_row - 1 >= 5 && cur_column - 1 >= 5 && dev_cell[(cur_row - 1)*num_per_row + cur_column - 1] == 3) {
			dev_cell[(cur_row - 1)*num_per_row + cur_column - 1] = 2;
			dev_medDir[(cur_row - 1)*num_per_row + cur_column - 1] = 0;
		}
		if (cur_column - 1 >= 5 && dev_cell[cur_row*num_per_row + cur_column - 1] == 3) {
			dev_cell[cur_row*num_per_row + cur_column - 1] = 2;
			dev_medDir[cur_row*num_per_row + cur_column - 1] = 0;
		}
		if (cur_row + 1 < WIDTH - 5 && cur_column - 1 >= 5 && dev_cell[(cur_row + 1)*num_per_row + cur_column - 1] == 3) {
			dev_cell[(cur_row + 1)*num_per_row + cur_column - 1] = 2;
			dev_medDir[(cur_row + 1)*num_per_row + cur_column - 1] = 0;
		}
		if (cur_row + 1 < WIDTH - 5 && dev_cell[(cur_row + 1)*num_per_row + cur_column] == 3) {
			dev_cell[(cur_row + 1)*num_per_row + cur_column] = 2;
			dev_medDir[(cur_row + 1)*num_per_row + cur_column] = 0;
		}
		if (cur_row + 1 < WIDTH - 5 && cur_column + 1 < HEIGHT - 5 && dev_cell[(cur_row + 1)*num_per_row + cur_column + 1] == 3) {
			dev_cell[(cur_row + 1)*num_per_row + cur_column + 1] = 2;
			dev_medDir[(cur_row + 1)*num_per_row + cur_column + 1] = 0;
		}
	}
	else if (dev_cell[cur_row*num_per_row + cur_column] == 3) {
		int direction = dev_medDir[cur_row*num_per_row + cur_column];
		if (direction == 1 && cur_column + 1 < HEIGHT) {
			medCellMoveCUDA(cur_row, cur_column, cur_row, cur_column + 1, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 2 && cur_row - 1 >= 5 && cur_column + 1 < HEIGHT - 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row - 1, cur_column + 1, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 3 && cur_row - 1 >= 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row - 1, cur_column, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 4 && cur_row - 1 >= 5 && cur_column - 1 >= 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row - 1, cur_column - 1, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 5 && cur_column - 1 >= 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row, cur_column - 1, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 6 && cur_row + 1 < WIDTH - 5 && cur_column - 1 >= 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row + 1, cur_column - 1, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 7 && cur_row + 1 < WIDTH - 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row + 1, cur_column, direction, dev_cell, dev_medDir, num_per_row);
		}
		else if (direction == 8 && cur_row + 1 < WIDTH - 5 && cur_column + 1 < HEIGHT - 5) {
			medCellMoveCUDA(cur_row, cur_column, cur_row + 1, cur_column + 1, direction, dev_cell, dev_medDir, num_per_row);
		}
	}
	else {
		dev_cell[cur_row*num_per_row + cur_column] = state;
	}
}

//Display individual pixels.
static void displayCUDA() {
	//bool host_ifDraw[WIDTH*HEIGHT] = { false };
	glClear(GL_COLOR_BUFFER_BIT);
	GLfloat red, green, blue;
	parallel_for(blocked_range2d<size_t>(0, WIDTH, 1000000, 0, HEIGHT, 1000000), [&](blocked_range2d<size_t> & r) {
		for (int i = r.rows().begin(); i != r.rows().end(); i++) {
			for (int j = r.cols().begin(); j != r.cols().end(); j++) {
				if (host_cell[i*HEIGHT + j] == 1) {
					red = 1;
					green = 0;
					blue = 0;
				}
				else if (host_cell[i*HEIGHT + j] == 2) {
					red = 0;
					green = 1;
					blue = 0;
				}
				else if (host_cell[i*HEIGHT + j] == 3) {
					red = 1;
					green = 1;
					blue = 0;
				}
				else {
					red = 0.0f;
					green = 0.0f;
					blue = 0.0f;
				}
				draw(red, green, blue, i, j);
			}
		}
		glutSwapBuffers();
	});
	// declare two array variables for devices and allocate memory for them
	int * dev_cell;
	int * dev_medDir;
	//bool * dev_ifDraw;
	gpuErrchk(hipMalloc((void**) &dev_cell, WIDTH*HEIGHT*sizeof(float)));
	gpuErrchk(hipMalloc((void**) &dev_medDir, WIDTH*HEIGHT*sizeof(float)));

	// copy cell and medDir array from host to device
	gpuErrchk(hipMemcpy(dev_cell, host_cell, WIDTH*HEIGHT*sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_medDir, host_medDir, WIDTH*HEIGHT * sizeof(float), hipMemcpyHostToDevice));

	cellHandle<<<1536,512>>>(dev_cell, dev_medDir, 1024, 768);
	
	gpuErrchk(hipMemcpy(host_cell, dev_cell, WIDTH*HEIGHT * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(host_medDir, dev_medDir, WIDTH*HEIGHT * sizeof(int), hipMemcpyDeviceToHost));

	gpuErrchk(hipFree(dev_cell));
	gpuErrchk(hipFree(dev_medDir));
}

int main(int argc, char** argv)
{
	int x = 1024;
	int y = 768;
	int mult = 1;

	tbb::task_scheduler_init tbb_init;
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
	glutInitWindowSize(WIDTH, HEIGHT);
	glutCreateWindow("Comp426 Assignment 3");
	init();
	setupCUDA(x, y, mult);
	glutDisplayFunc(displayCUDA);
	glutTimerFunc(0, update, 0);
	glutMouseFunc(OnMouseClick);
	glutMainLoop();
	return 0;
}
